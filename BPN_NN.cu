#include "hip/hip_runtime.h"
/*
 * File:   BPN_NN.cu
 * Author: Cynthia Castillo
 * Student ID: A01374530
 *
 */

#include <iostream>
#include <cstdio>
#include <chrono>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <algorithm>
#include <cmath>
#include <random>

#include "common.h"
#include <hip/hip_runtime.h>

__device__ float sigmoidalGradiente(float z) {
  float g_z = 1.0 / (1.0 + std::exp(-z));

  return g_z * (1 - g_z);
}

__global__ void matrixMult(float *A, float *B, float *C, const int A_rows, const int B_cols)
{
    unsigned int ix_rows = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy_cols = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy_cols * A_rows + ix_rows;

    if (ix_rows < A_rows && iy_cols < B_cols)
    {
    	C[idx] = 0;
    	for (int shared_dim = 0; shared_dim < A_rows; shared_dim++)
    		//dot product
    		C[idx] += A[shared_dim * B_cols + ix_rows] * B[iy_cols * B_cols + shared_dim];  

    	C[idx] = sigmoidalGradiente(C[idx]);
    }
}

__global__ void transpose(float *A, float *C, const int rows, const int cols)
{
    unsigned int ix_rows = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy_cols = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = ix_rows * rows + iy_cols;
    unsigned int idy = iy_cols * cols + ix_rows;

    if (ix_rows < cols && iy_cols < rows)
    	C[idy] = A[idx];
}

__global__ void elemWise(float *A, float *B, float *C, const int opt, const int rows, const int cols)
{
    unsigned int ix_rows = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy_cols = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy_cols * rows + ix_rows;

    if (ix_rows < rows && iy_cols < cols)
    {
    	if (opt == '-')
    		C[idx] = A[idx] - B[idx];

    	else if (opt == '*')
    		C[idx] = A[idx] * B[idx];

    	else if (opt == 'm')
    		C[idx] = B[1 * rows + ix_rows];
    }
}

__global__ void costFunc(float *A, float *Y, float *cost, const int rows, const int cols)
{
    unsigned int ix_rows = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy_cols = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy_cols * rows + ix_rows;

    if (ix_rows < rows && iy_cols < cols)
    	cost[0] += (-Y[idx] * log(A[idx])) - ((1 - Y[idx])*log(1 - A[idx]));
}

__global__ void cumulativeSum(float *A, float *result, const int rows, const int cols)
{
    unsigned int ix_rows = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy_cols = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy_cols * rows + ix_rows;

    if (ix_rows < rows && iy_cols < cols)
    	result[iy_cols] += A[idx];

    /* INTENTO DE HACER CON SHARED MEMORY
    int sumIndex_X = threadIdx.x;
    extern __shared__ float s_sum[];


    if (ix_rows < rows && iy_cols < cols)
    	s_sum[sumIndex_X] = A[idx];

     __syncthreads();

    // For reductions, threadsPerBlock must be a power of 2
    int i = blockDim.x / 2;
    while(i != 0)
    {
        if(sumIndex_X < i)
            s_sum[sumIndex_X] += s_sum[sumIndex_X + i];
        __syncthreads();
        i /= 2;
    }

    
    if (iy_cols < cols)
    {
    	if(sumIndex_X == 0)
        	result[blockIdx.y] = s_sum[0];
    }
    */
}

int main(int argc, char *argv[])
{
	//Parameters
	
	float sigma = 0.12; //weights range value
	int training_size = 1; // Number of examples
	int training_attr_size = 400; // number of features (400 values (20 x 20 pixels))
	int hidden_layer_size = 25;
	int output_layer_size = 10; //10 digits

	std::string filePath;
	
	if(argc < 3){
		filePath = "nueve.txt";
		training_size = 1;
	}
  	else{
  	  	filePath = argv[1];
  	  	training_size = std::stoi(argv[2]);
  	}

  	std::cout << "\nFile used: " << filePath <<  "\nNumber of examples: " << training_size << std::endl;


  	int lineCounter = 0;
  	// FILE READER
    int counter = 0;
	std::ifstream file(filePath);
	std::string line;
	char split_char = ' ';

	std::vector<std::string> values_s;
	std::vector<std::string> labels_s;

	while (std::getline(file, line))
	{
		counter = 0;

		std::istringstream split(line);
		for (std::string each; std::getline(split, each, split_char); counter <= training_attr_size ? values_s.push_back(each) : labels_s.push_back(each))
			counter++;

		lineCounter++;
	    //std::cout << "Counter: " << lineCounter << std::endl;

	    if (lineCounter == training_size)
	      break;
	}

	// TRANSFORM DATA TO float
	std::vector<float> values(values_s.size());
	std::vector<int> labels(labels_s.size());

	std::transform(values_s.begin(), values_s.end(), values.begin(), [](const std::string& val)
	{
	    return std::stod(val);
	});

	std::transform(labels_s.begin(), labels_s.end(), labels.begin(), [](const std::string& val)
	{
	    return std::stoi(val);
	});

	// GENERATE WEIGHTS
	std::random_device rand_dev;
  	std::default_random_engine generator(rand_dev());
	std::uniform_real_distribution<float> distribution(-sigma, sigma);

	std::vector<float> w1;
	std::vector<float> w2;

	//Initialize W1
	for(int i = 0; i < training_attr_size * hidden_layer_size; i++)
		w1.push_back(distribution(generator));

	//Initialize W2
	for(int i = 0; i < hidden_layer_size * output_layer_size; i++)
		w2.push_back(distribution(generator));


	// Z2 will be the Results matrix
	float *h_values, *h_labels, *h_w1, *h_w2, *h_z2, *h_outputs, *h_J;
  	h_values = (float *)malloc(values.size() * sizeof(float));
	h_labels = (float *)malloc(labels.size() * sizeof(float));
	h_w1 = (float *)malloc(w1.size() * sizeof(float));
	h_w2 = (float *)malloc(w2.size() * sizeof(float));
	h_z2 = (float *)calloc(training_size * output_layer_size, sizeof(float));
	h_outputs = (float *)calloc(training_size * output_layer_size, sizeof(float));
	h_J = (float *)calloc(1, sizeof(float));
	for (int i = 0; i < training_size; i++)
		h_outputs[(labels[i]-1) * training_size + i] = 1;

	h_values = values.data();
	h_w1 = w1.data();
	h_w2 = w2.data();

	float *d_values, *d_labels, *d_w1, *d_w2, *d_z1, *d_z2, *d_b1, *d_b2, *d_outputs, *d_delta3, *d_delta2, *d_delta1, *d_transW2, *d_transW1, *d_J, *d_delta2Sum, *d_delta1Sum;
	SAFE_CALL(hipMalloc((void **)&d_values, values.size() * sizeof(float)), "Error allocating d_values");
	SAFE_CALL(hipMalloc((void **)&d_labels, labels.size() * sizeof(float)), "Error allocating d_labels");
	SAFE_CALL(hipMalloc((void **)&d_w1, w1.size() * sizeof(float)), "Error allocating d_w1");
	SAFE_CALL(hipMalloc((void **)&d_w2, w2.size() * sizeof(float)), "Error allocating d_w2");
	SAFE_CALL(hipMalloc((void **)&d_z1, training_size * hidden_layer_size * sizeof(float)), "Error allocating d_z1");
	SAFE_CALL(hipMalloc((void **)&d_z2, training_size * output_layer_size * sizeof(float)), "Error allocating d_z2");
	SAFE_CALL(hipMalloc((void **)&d_b1, training_attr_size * sizeof(float)), "Error allocating d_b1");
	SAFE_CALL(hipMalloc((void **)&d_b2, hidden_layer_size * sizeof(float)), "Error allocating d_b2");
	SAFE_CALL(hipMalloc((void **)&d_outputs, training_size * output_layer_size * sizeof(float)), "Error allocating d_outputs");
	SAFE_CALL(hipMalloc((void **)&d_delta3, training_size * output_layer_size * sizeof(float)), "Error allocating d_delta3");
	SAFE_CALL(hipMalloc((void **)&d_delta2, training_size * hidden_layer_size * sizeof(float)), "Error allocating d_delta2");
	SAFE_CALL(hipMalloc((void **)&d_transW2, output_layer_size * hidden_layer_size * sizeof(float)), "Error allocating d_transW2");
	SAFE_CALL(hipMalloc((void **)&d_J, 1 * sizeof(float)), "Error allocating d_J");
	SAFE_CALL(hipMalloc((void **)&d_delta2Sum, hidden_layer_size * sizeof(float)), "Error allocating d_delta2Sum");
	SAFE_CALL(hipMalloc((void **)&d_delta1, training_size * training_attr_size * sizeof(float)), "Error allocating d_delta1");
	SAFE_CALL(hipMalloc((void **)&d_delta1Sum, training_attr_size * sizeof(float)), "Error allocating d_delta1Sum");
	SAFE_CALL(hipMalloc((void **)&d_transW1, hidden_layer_size * training_attr_size * sizeof(float)), "Error allocating d_transW2");

	// transfer data from host to device
	SAFE_CALL(hipMemcpy(d_values, h_values, values.size() * sizeof(float), hipMemcpyHostToDevice), "Error copying d_values");
	SAFE_CALL(hipMemcpy(d_labels, h_labels, labels.size() * sizeof(float), hipMemcpyHostToDevice), "Error copying d_labels");
	SAFE_CALL(hipMemcpy(d_w1, h_w1, w1.size() * sizeof(float), hipMemcpyHostToDevice), "Error copying d_w1");
	SAFE_CALL(hipMemcpy(d_w2, h_w2, w2.size() * sizeof(float), hipMemcpyHostToDevice), "Error copying d_w2");
	SAFE_CALL(hipMemcpy(d_z2, h_z2, training_size * output_layer_size * sizeof(float), hipMemcpyHostToDevice), "Error copying d_z2");
	SAFE_CALL(hipMemcpy(d_outputs, h_outputs, training_size * output_layer_size * sizeof(float), hipMemcpyHostToDevice), "Error copying d_outputs");
	hipMemset(d_delta3, 0, training_size * output_layer_size*sizeof(float));
	hipMemset(d_delta2, 0, training_size * hidden_layer_size*sizeof(float));
	hipMemset(d_transW2, 0, output_layer_size * hidden_layer_size*sizeof(float));
	hipMemset(d_J, 0, 1*sizeof(float));
	hipMemset(d_delta2Sum, 0, hidden_layer_size*sizeof(float));
	hipMemset(d_delta1, 0, training_size * training_attr_size *sizeof(float));
	hipMemset(d_transW1, 0, hidden_layer_size * training_attr_size *sizeof(float));
	hipMemset(d_delta1Sum, 0, training_attr_size *sizeof(float));
    
	// INVOKE KERNEL
	int dimx = 32;
	int dimy = 32;
	dim3 block(dimx, dimy);
	dim3 grid((training_attr_size + block.x - 1) / block.x, (training_size + block.y - 1) / block.y);

	//FORWARD PROP
	std::cout << "\n *** Forward Propagation *** \n";
	std::chrono::duration<float, std::milli> total_time = std::chrono::high_resolution_clock::duration::zero();
	std::chrono::duration<float, std::milli> duration_ms = std::chrono::high_resolution_clock::duration::zero();
	auto start_gpu =  std::chrono::high_resolution_clock::now();
 	matrixMult<<<grid, block>>>(d_values, d_w1, d_z1, training_size, hidden_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 1");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    auto end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
	std::cout << "Matrix Mult (Values x W1) time: " << duration_ms.count() << "\n";
	start_gpu =  std::chrono::high_resolution_clock::now();
 	matrixMult<<<grid, block>>>(d_z1, d_w2, d_z2, training_size, output_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 2");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
	std::cout << "Matrix Mult (Z1 x W2) time: " << duration_ms.count() << "\n";

    //Calculating Cost
    std::cout << "\n *** Calculating Cost *** \n";
    start_gpu =  std::chrono::high_resolution_clock::now();
    costFunc<<<grid, block>>>(d_z2, d_outputs, d_J, training_size, output_layer_size);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel Cost");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
	std::cout << "Cost calcultation time: " << duration_ms.count() << "\n";

    //SAFE_CALL(hipMemcpy(h_J, d_J, 1 * sizeof(float), hipMemcpyDeviceToHost), "CUDA Memcpy Device To Host Failed");
    //std::cout << "Costo: " << h_J[0] << std::endl;

    //BACKPROPAGATION
    std::cout << "\n *** BackPropagation *** \n";
    //Delta 3
    start_gpu =  std::chrono::high_resolution_clock::now();
 	elemWise<<<grid, block>>>(d_z2, d_outputs, d_delta3, '-', training_size, output_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 3");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
	std::cout << "Element wise operation time: " << duration_ms.count() << "\n";
	start_gpu =  std::chrono::high_resolution_clock::now();
	//Delta 2
    transpose<<<grid, block>>>(d_w2, d_transW2, hidden_layer_size, output_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 4");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
	std::cout << "Transpose time: " << duration_ms.count() << "\n";
	start_gpu =  std::chrono::high_resolution_clock::now();
    matrixMult<<<grid, block>>>(d_delta3, d_transW2, d_delta2, training_size, hidden_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 5");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
    start_gpu =  std::chrono::high_resolution_clock::now();
    elemWise<<<grid, block>>>(d_delta2, d_z1, d_delta2, '*', training_size, hidden_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 6");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
    start_gpu =  std::chrono::high_resolution_clock::now();
    cumulativeSum<<<grid, block>>>(d_delta2, d_delta2Sum, training_size, hidden_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 7");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
	std::cout << "Cummulative Summation time: " << duration_ms.count() << "\n";
	//Delta 1
	start_gpu =  std::chrono::high_resolution_clock::now();
    transpose<<<grid, block>>>(d_w1, d_transW1, training_attr_size, hidden_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 8");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
    matrixMult<<<grid, block>>>(d_delta2, d_transW1, d_delta1, training_size, training_attr_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 9");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    start_gpu =  std::chrono::high_resolution_clock::now();
    cumulativeSum<<<grid, block>>>(d_delta1, d_delta1Sum, training_size, training_attr_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 10");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;

    //Updating Weights
    std::cout << "\n *** Weights update ***\n";
    start_gpu =  std::chrono::high_resolution_clock::now();
    elemWise<<<grid, block>>>(d_w2, d_delta2Sum, d_w2, 'm', hidden_layer_size, output_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 10");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
	std::cout << "W2 update time: " << duration_ms.count() << "\n";
	start_gpu =  std::chrono::high_resolution_clock::now();
    elemWise<<<grid, block>>>(d_w1, d_delta1Sum, d_w1, 'm', training_attr_size, hidden_layer_size);
 	SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel 11");
    SAFE_CALL(hipGetLastError(), "Error with last error");
    end_gpu =  std::chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	total_time = total_time + duration_ms;
	std::cout << "W1 update time: " << duration_ms.count() << "\n";

	std::cout << "\nTotal time: " << total_time.count() << "\n";


	return 0;	
}


 	//float *h_w3;
	//h_w3 = (float *)malloc(output_layer_size * hidden_layer_size * sizeof(float));
	//SAFE_CALL(hipMemcpy(h_w3, d_transW2, output_layer_size * hidden_layer_size * sizeof(float), hipMemcpyDeviceToHost), "CUDA Memcpy Device To Host Failed");
    
 	//matrixMult<<<grid, block>>>(d_w2, d_z1, d_z2, output_layer_size, training_size, hidden_layer_size);

 	//SAFE_CALL(hipMemcpy(h_z2, d_z2, z2.size(), hipMemcpyDeviceToHost), "CUDA Memcpy Device to host Failed");



   	/*
   		  std::cout << " ** W1 ** " << std::endl;
  counter = 0;
  for (auto i = w1.begin(); i != w1.end(); ++i){
    std::cout << *i << ' ';
  counter++;
  if (counter%hidden_layer_size == 0)
    std::cout << std::endl;
  }
  std::cout << std::endl << std::endl;
	
	counter = 0;
	std::cout << " ** Values ** " << std::endl;
	for (auto i = values.begin(); i != values.end(); ++i){
    std::cout << *i << ' ';
    counter++;
	if (counter%training_attr_size == 0)
		std::cout << std::endl << std::endl;
	}
	std::cout << std::endl << std::endl;
	

    for (int i = 0; i <  training_size * hidden_layer_size; i++){
    	if (i%hidden_layer_size == 0)
    		std::cout << std::endl;
		std::cout << h_z1[i] << " ";
    }
    std::cout << std::endl << std::endl;
    */